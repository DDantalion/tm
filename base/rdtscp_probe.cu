#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <x86intrin.h>

#define BUFFER_SIZE (64 * 1024 * 1024)  // 64MB
#define ITERATIONS 10000

#define CHECK(call) \
    if ((call) != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(call) << std::endl; \
        exit(EXIT_FAILURE); \
    }

__global__ void migrate_kernel(char *buf, size_t size) {
    for (size_t j = 0; j < 600; ++j) {
        for (size_t i = 0; i < size; i += size / sizeof(char)) {
            buf[i] += 1;
            if (buf[i] > 100) buf[i] -= 5;
        }
    }
}

int main() {
    int local_gpu = 0;
    int remote_gpu = 1;
    CHECK(hipSetDevice(local_gpu));

    char *buf = nullptr;
    CHECK(hipMallocManaged(&buf, BUFFER_SIZE));

    // Prefetch to remote GPU (simulate remote residency)
    CHECK(hipMemAdvise(buf, BUFFER_SIZE, hipMemAdviseSetPreferredLocation, remote_gpu));
    CHECK(hipMemAdvise(buf, BUFFER_SIZE, hipMemAdviseSetAccessedBy, local_gpu));
    CHECK(hipMemPrefetchAsync(buf, BUFFER_SIZE, remote_gpu));  // ensure remote residency
    CHECK(hipDeviceSynchronize());

    for (int i = 0; i < ITERATIONS; ++i) {
        unsigned aux;
        uint64_t start = __rdtscp(&aux);
        migrate_kernel<<<1, 1>>>(buf, BUFFER_SIZE);
        CHECK(hipDeviceSynchronize());
        uint64_t end = __rdtscp(&aux);
        std::cout << "Cycle: " << (end - start) << std::endl;
    }

    CHECK(hipFree(buf));
    return 0;
}
