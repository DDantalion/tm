#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

#define BUFFER_SIZE (64 * 1024 * 1024)
#define CHECK(call) \
    if ((call) != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(call) << std::endl; \
        exit(EXIT_FAILURE); \
    }

__global__ void migrate_kernel(char *buf, size_t size) {
    for (size_t j = 0; j < 600; ++j) {
        for (size_t i = 0; i < size; i += size / sizeof(char)) {
            buf[i] += 1;
            if (buf[i] > 100) buf[i] -= 5;
        }
    }
}

int main() {
    int local_gpu = 1;
    int remote_gpu = 0;
    CHECK(hipSetDevice(local_gpu));

    char *buf = nullptr;
    CHECK(hipMallocManaged(&buf, BUFFER_SIZE));

    CHECK(hipMemAdvise(buf, BUFFER_SIZE, hipMemAdviseSetPreferredLocation, remote_gpu));
    CHECK(hipMemAdvise(buf, BUFFER_SIZE, hipMemAdviseSetAccessedBy, local_gpu));
    CHECK(hipMemPrefetchAsync(buf, BUFFER_SIZE, remote_gpu));
    CHECK(hipDeviceSynchronize());

    for (int i = 0; i < 100; ++i) {
        migrate_kernel<<<1, 1>>>(buf, BUFFER_SIZE);
        CHECK(hipDeviceSynchronize());
    }

    std::cout << "Program B completed remote page migrations.\n";
    CHECK(hipFree(buf));
    return 0;
}
