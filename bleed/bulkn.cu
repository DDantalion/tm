#include <hip/hip_runtime.h>
#include <iostream>


#define CHECK(call) \
    if ((call) != hipSuccess) { \
        std::cerr << "CUDA error: " << hipGetErrorString(call) << std::endl; \
        exit(EXIT_FAILURE); \
    }

int main(int argc, char **argv) {
    int dev0 = 1, dev1 = 0;
    size_t TRANSFER_SIZE = 256;
    size_t count = 1000;

    for (int i = 1; i < argc; ++i) {
        if (strcmp(argv[i], "--size") == 0 && i + 1 < argc) TRANSFER_SIZE = atol(argv[++i]);
        if (strcmp(argv[i], "--count") == 0 && i + 1 < argc) count = atol(argv[++i]);
    }
    CHECK(hipSetDevice(dev0));

    char *src, *dst;
    CHECK(hipMalloc(&src, TRANSFER_SIZE));
    CHECK(hipSetDevice(dev1));
    CHECK(hipMalloc(&dst, TRANSFER_SIZE));

    CHECK(hipSetDevice(dev0));
    for (int i = 0; i < count; ++i) {
        CHECK(hipMemcpyPeer(dst, dev1, src, dev0, TRANSFER_SIZE));
        CHECK(hipDeviceSynchronize());
    }

    std::cout << "Program B: Completed 100 transfers of 16MB." << std::endl;
    return 0;
}
