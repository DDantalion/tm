#include <hip/hip_runtime.h>
#include <iostream>

#define TRANSFER_SIZE (16 * 1024 * 1024)
#define CHECK(call) \
    if ((call) != hipSuccess) { \
        std::cerr << "CUDA error: " << hipGetErrorString(call) << std::endl; \
        exit(EXIT_FAILURE); \
    }

int main() {
    int dev0 = 0, dev1 = 1;
    CHECK(hipSetDevice(dev0));

    char *src, *dst;
    CHECK(hipMalloc(&src, TRANSFER_SIZE));
    CHECK(hipSetDevice(dev1));
    CHECK(hipMalloc(&dst, TRANSFER_SIZE));

    CHECK(hipSetDevice(dev0));
    for (int i = 0; i < 100; ++i) {
        CHECK(hipMemcpyPeer(dst, dev1, src, dev0, TRANSFER_SIZE));
        CHECK(hipDeviceSynchronize());
    }

    std::cout << "Program B: Completed 100 transfers of 16MB." << std::endl;
    return 0;
}
