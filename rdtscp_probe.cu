#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <x86intrin.h>
#include <cstring>

#define CHECK(call) \
    if ((call) != hipSuccess) { \
        std::cerr << "CUDA error: " << hipGetErrorString(call) << std::endl; \
        exit(EXIT_FAILURE); \
    }

__global__ void migrate_kernel(char *buf, size_t size) {
    for (size_t j = 0; j < 600; ++j) {
        for (size_t i = 0; i < size; i += size / sizeof(char)) {
            buf[i] += 1;
            if (buf[i] > 100) buf[i] -= 5;
        }
    }
}


int main(int argc, char** argv) {
    size_t size = 256 * 1024 * 1024;
    int freq = 10000;

    for (int i = 1; i < argc; ++i) {
        if (strcmp(argv[i], "--freq") == 0 && i + 1 < argc) freq = atoi(argv[++i]);
        if (strcmp(argv[i], "--size") == 0 && i + 1 < argc) size = atol(argv[++i]);
    }

    int local_gpu = 0, remote_gpu = 1;
    CHECK(hipSetDevice(local_gpu));

    char *buf;
    CHECK(hipMallocManaged(&buf, size));
    CHECK(hipMemAdvise(buf, size, hipMemAdviseSetPreferredLocation, remote_gpu));
    CHECK(hipMemAdvise(buf, size, hipMemAdviseSetAccessedBy, local_gpu));
    CHECK(hipMemPrefetchAsync(buf, size, remote_gpu));
    CHECK(hipDeviceSynchronize());

    for (int i = 0; i < freq; ++i) {
        unsigned aux;
        uint64_t start = __rdtscp(&aux);
        migrate_kernel<<<1, 1>>>(buf, size);
        CHECK(hipDeviceSynchronize());
        uint64_t end = __rdtscp(&aux);
        std::cout << "Cycle: " << (end - start) << std::endl;
    }

    CHECK(hipFree(buf));
    return 0;
}
