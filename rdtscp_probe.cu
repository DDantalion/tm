#include <hip/hip_runtime.h>
#include <iostream>
#include <x86intrin.h>

#define TRANSFER_SIZE 256  // bytes
#define ITERATIONS 1000000
#define CHECK(call) \
    if ((call) != hipSuccess) { \
        std::cerr << "CUDA error: " << hipGetErrorString(call) << std::endl; \
        exit(EXIT_FAILURE); \
    }

int main() {
    int dev0 = 0, dev1 = 1;
    CHECK(hipSetDevice(dev0));

    char *src, *dst;
    CHECK(hipMalloc(&src, TRANSFER_SIZE));
    CHECK(hipSetDevice(dev1));
    CHECK(hipMalloc(&dst, TRANSFER_SIZE));

    CHECK(hipSetDevice(dev0));

    for (int i = 0; i < ITERATIONS; ++i) {
        unsigned int aux;
        uint64_t start = __rdtscp(&aux);
        CHECK(hipMemcpyPeer(dst, dev1, src, dev0, TRANSFER_SIZE));
        CHECK(hipDeviceSynchronize());
        uint64_t end = __rdtscp(&aux);
        std::cout << "Cycle: " << (end - start) << std::endl;
    }

    return 0;
}
